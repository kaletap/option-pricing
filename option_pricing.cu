#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <hiprand.h>
#include <iostream>

#define S_0 98
#define TIME 1
#define SIGMA 0.2
#define R 0.05
#define N_MAX 50000
#define K 100
//#define double float
#define DEFAULT_N 50000

using namespace std;

struct OptionPrice {
    double r;
    double v;
    double t;
    double s;
    double k;
    OptionPrice(
        double _underlying_price, 
        double _interest_rate, 
        double _sigma, 
        double _time_to_expiry,
        double _strike_price) {
            s = _underlying_price;
            r = _interest_rate;
            v = _sigma;
            t = _time_to_expiry;
            k = _strike_price;
    }
    __device__ double operator()(const double &std_normal_variable) const {
        double asset_price 
            = s * exp((r - 0.5 * v*v) * t + v*sqrt(t)*std_normal_variable);
        return exp(-r*t) * max(0.0, asset_price - k);
    }
};

struct SquaredError {
    double mean;
    SquaredError(const double _mean) {
        mean = _mean;
    }
    __host__ __device__ double operator()(const double x) const {
        return (x - mean)*(x - mean);
    }
};

int main(int argc, char *argv[]) {
    size_t n;
    if (argc > 1) {
        n = atoi(argv[1]);
    }
    else {
        n = DEFAULT_N;
    }
	// Allocate n doubles on host
    thrust::device_vector<double> d_data(n, 0);

    // Creating CURAND generator
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT); 
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ll);

    // Generate points from random distribution
    double *d_data_ptr = thrust::raw_pointer_cast(&d_data[0]);
    hiprandGenerateNormalDouble(gen, d_data_ptr, n, 0.0, 1.0);

    // Calculate 
    OptionPrice option_price(S_0, R, SIGMA, TIME, K);
    thrust::transform(d_data.begin(), d_data.end(), d_data.begin(), option_price);
    double sum =
        thrust::reduce(d_data.begin(), d_data.end(), 0.0, thrust::plus<double>());
    double mean = sum / n;
    double squared_error =  
        thrust::transform_reduce(d_data.begin(), d_data.end(),
            SquaredError(mean), 0.0, thrust::plus<double>());
    double standard_deviation = sqrt(squared_error / n - 1);

    cout << "First 10 profits:" << endl;
    for (int i = 0; i < 10; ++i) {
        cout << d_data[i] << " ";
    }
    cout << endl;

    cout << "Proft mean of " << n << " observations is " 
        << mean << " with standard deviation of " << standard_deviation << endl;

    return 0;
}
